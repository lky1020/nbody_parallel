#include "hip/hip_runtime.h"
﻿#include "Kernels.cuh"
#include "Util.h"

#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>

#include "Constants.h"
#include "Simulation.h"

/* Variables that used to store the position at the global memory to have a faster access to the body's position*/
__shared__ float3 posShare[THREAD_NUM];

int main() {
    /*
    * A SIMPLE PROCESS OF N BODY SIMULATION IN SERIAL
    *
    * STEP	DESCRIPTION
    * ---------------------
    *  1.	INITIALIZE THE WINDOW WIDTH AND HEIGHT
    *  2.	RANDOM GENERATE THE BODY BASED ON THE NUMBER OF BODIES	(BY DEFAULT NUM_BODIES = 1024)
    *  3.	SET THE ANGLE, RADIUS, VELOCITY OF THE BODY GENERTATE AND PLACE
    *  4.	PLACE A BODY AT THE CENTER (ACT AS A SUN) BY ASSIGNING A HEAVIEST VALUE OF MASS
    *  5.	A LOOP WITH i =  1023 (MINUS 1 BCOZ OF SUN) IS TAKE PLACE FOR PROCESS 3 AND 4
    *  6.	CREATE AND SET WINDOW AND VIEW
    *  7.	START THE STIMULATION
    *  8.	CHECK THE SFML EVENT (WINDOW EVENT, MOUSE SCROLL AND KEY PRESS)
    *  9.	UPDATE THE BODIES BY CALCULATING THE EFFECTS OF INTERACTION (POSITION, DISTANCE, FORCE AND ACCELERATION) BETWEEN 2 BODIES
    * 10.	UPDATE BODY VELOCITY, POSITION AND ACCELERATION
    * 11.	PRESENT THE BODIES IN WINDOWS
    * ***	ALL THE BODIES EXCEPT SUN (THE BODY AT THE CENTER) HAVING THE EQUAL VALUE OF MASS
    * ***	LIBRARY SFML IS USED TO DISPLAY BODIES
    * ***	TWO MAIN CLASS WHICH HANDLE DEFINES THE BODY (body.cpp) AND CARRY OUT SIMULATION (simulation.cpp)
    */


    /*
    * Initialize the windows size and generate random bodies
    */
    Simulation nBody_sim(WIDTH, HEIGHT);

    /*
    * Start Stimulation
    */
    nBody_sim.start();

    return EXIT_SUCCESS;
}

/* CUDA memory allocations and copy memory to the GPU*/
void updateInCUDA(std::vector<Body>& bodies_h, int nBodies, int nThreads) {

    /* Declaration for necessary variables needed */
        /* Number of bytes required for bodies*/
    int size;

    /* Number of Blocks */
    int nBlocks;
    /* Buffer for bodies */
    Body* bodies_d;

    /* Initialization */
        /* Dynamically allocate host memory */
    size = sizeof(Body) * nBodies;
    /* Number of Blocks */
    nBlocks = nBodies / nThreads;

    /* Start CUDA Memory Allocation and Copy Memory to CPU */
        /* Allocate device Memory */
    hipMalloc((void**)&bodies_d, size);

    /* Copy Host Memory to Device Memory */
    hipMemcpy(bodies_d, &bodies_h[0], size, hipMemcpyHostToDevice);

    /* Launch Kernel */
    /* Called a CUDA kernel with <nBlocks> block and that one block has <THREAD_NUM> active threads.*/
    interactAndUpdate << < nBlocks, nThreads >> > (bodies_d);

    /* Synchronize */
    /* Forces the program to wait for all previously issued commands in
    *  all streams on the device to finish before continuing (from the CUDA C Programming Guide).
    *  So when GPU device is executing kernel, the CPU can continue to work on some other commands and
    *  issue more instructions to the device
    */
    hipDeviceSynchronize();

    /* Copy results to host */
    /* Straight away overwrite the original vector */
    hipMemcpy(&bodies_h[0], bodies_d, size, hipMemcpyDeviceToHost);

    /* Cleanup */
    hipFree(bodies_d);
}

/* This function calculate the force of the body and update the velocity,
*  position and acceleration of the body.
*/
__global__ void interactAndUpdate(Body* bodies) {

    /* Call function calculate the effects of an interaction between 2 bodies */
    accumulate(bodies);
}

__device__ void accumulate(Body* bodies) {


    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    extern __shared__ float3 posShare[THREAD_NUM];


    for (int tile = 0; tile < gridDim.x; tile++) {
        /* Assign the value of the body position into the shared position */
        auto tpos = bodies[tile * blockDim.x + threadIdx.x].position();
        posShare[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);

        /* To ensure all the threads reached before proceed to the following process */
        __syncthreads();

        /* Update the Body interaction (Get from the serial code) */
        for (int j = idx + 1; j < THREAD_NUM; ++j) {
            if (idx != j) {
                // vector to store the position difference between the 2 bodies
                vec3 posDiff{};
                posDiff.x = (posShare[j].x - bodies[idx].position().x) * TO_METERS;
                posDiff.y = (posShare[j].y - bodies[idx].position().y) * TO_METERS;
                posDiff.z = (posShare[j].z - bodies[idx].position().z) * TO_METERS;
                // the actual distance is the length of the vector
                auto dist = sqrtf(posDiff.x * posDiff.x + posDiff.y * posDiff.y +
                    posDiff.z * posDiff.z);
                // calculate force
                double F = TIME_STEP * (G * bodies[idx].mass() * bodies[j].mass()) /
                    ((dist * dist + SOFTENING * SOFTENING) * dist);

                // set this body's acceleration
                bodies[j].acceleration().x += F * posDiff.x / bodies[j].mass();
                bodies[j].acceleration().y += F * posDiff.y / bodies[j].mass();
                bodies[j].acceleration().z += F * posDiff.z / bodies[j].mass();


            }

        }

    }

    /* To ensure all the threads reached before proceed to the following process */
    __syncthreads();

    /* Update velocity */
    bodies[idx].velocity().x += bodies[idx].acceleration().x;
    bodies[idx].velocity().y += bodies[idx].acceleration().y;
    bodies[idx].velocity().z += bodies[idx].acceleration().z;

    /* reset acceleration */
    bodies[idx].acceleration().x = 0.0;
    bodies[idx].acceleration().y = 0.0;
    bodies[idx].acceleration().z = 0.0;

    /* update position */
    bodies[idx].position().x += TIME_STEP * bodies[idx].velocity().x / TO_METERS;
    bodies[idx].position().y += TIME_STEP * bodies[idx].velocity().y / TO_METERS;
    bodies[idx].position().z += TIME_STEP * bodies[idx].velocity().z / TO_METERS;

}
